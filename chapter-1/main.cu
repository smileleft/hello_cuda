#include "hip/hip_runtime.h"
#include <cstdio>

dim3 threadsPerBlock(16, 16);
dim3 blocks((N + threadsPerBlock.x - 1) / threadsPerBlock.x, (M + threadsPerBlock.y - 1) / threadsPerBlock.y);

 
__global__ void increment_kernel(int* d_val) {
    // 모든 스레드가 d_val[0]을 1 증가
    d_val[0] += 1;
}

__global__ void vector_add_kernel(const float* a, const float* b, float* c, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x; 
    if (idx < N) {
        c[idx] = a[idx] + b[idx];
    }
}
 
int main() {
    int h_val = 10;          // Host 메모리에 있는 정수
    int* d_val = nullptr;    // Device 메모리 포인터
 
    // Device 메모리 할당
    hipMalloc(&d_val, sizeof(int));
    // Host -> Device 복사
    hipMemcpy(d_val, &h_val, sizeof(int), hipMemcpyHostToDevice);
 
    // GPU 커널 런칭(블록 1개, 스레드 1개)
    increment_kernel<<<1,1>>>(d_val);
    // GPU 완료 대기
    hipDeviceSynchronize();
 
    // Device -> Host 복사 (결과 가져오기)
    hipMemcpy(&h_val, d_val, sizeof(int), hipMemcpyDeviceToHost);
 
    printf("Result: %d\n", h_val); // 원래 10에서 +1 증가 -> 11 예상
 
    hipFree(d_val);
    return 0;
}
