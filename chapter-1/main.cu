
#include <hip/hip_runtime.h>
#include <cstdio>
 
// GPU에서 실행되는 커널 함수
__global__ void hello_kernel() {
    printf("Hello from the GPU!\\n");
}
 
int main() {
    // GPU 커널 런칭: 1개의 블록, 1개의 스레드
    hello_kernel<<<1,1>>>();
    // GPU 연산이 완료될 때까지 동기화
    hipDeviceSynchronize();
    return 0;
}
